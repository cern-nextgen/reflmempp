#include "hip/hip_runtime.h"
#include <benchmark/benchmark.h>
#include <chrono>
#include <iostream>
#include <numeric>
#include <span>
#include <vector>

#include <chrono>
using Clock = std::chrono::steady_clock;
using fsecs = std::chrono::duration<double, std::chrono::seconds::period>;

#define CHECK_CUDA_ERROR(call)                                                                                         \
  do {                                                                                                                 \
    hipError_t err = call;                                                                                            \
    if (err != hipSuccess) {                                                                                          \
      fprintf(stderr, "CUDA error in file '%s' in line %i: %s.\n", __FILE__, __LINE__, hipGetErrorString(err));       \
      exit(EXIT_FAILURE);                                                                                              \
    }                                                                                                                  \
  } while (0)

constexpr size_t container_size = 2 << 23;

#ifdef __HIPCC__
#define __HOST_DEVICE__ __host__ __device__
#else
#define __HOST_DEVICE__
#endif

// #define USE_CUDA_EVENTS

// https://github.com/cern-nextgen/wp1.7-soa-wrapper/blob/0aa71e432691425353ea3c4ba7cd6f51f480746c/allocator.h#L10

enum class layout { aos = 0, soa = 1 };
enum class location { host = 0, device = 1 };

template <typename T, location D>
struct SRef;

template <typename T>
struct SRef<T, location::host> {
  std::span<T> x, y, z;

  struct Ref {
    T &x, &y, &z;
  };

  struct RefConst {
    const T &x, &y, &z;
  };

  constexpr static size_t alignment = 128;

  Ref operator[](size_t i) { return {x[i], y[i], z[i]}; }

  const RefConst operator[](size_t i) const { return {x[i], y[i], z[i]}; }

  // Helper function to compute aligned size
  constexpr inline size_t align_size(size_t size) const { return ((size + alignment - 1) / alignment) * alignment; }

  SRef(size_t n) {
    storage.resize(align_size(3 * n * sizeof(T)));
    new (storage.data()) T[n];

    x = std::span<T>(reinterpret_cast<T *>(storage.data()), n);
    y = std::span<T>(reinterpret_cast<T *>(storage.data() + align_size(n * sizeof(T))), n);
    z = std::span<T>(reinterpret_cast<T *>(storage.data() + align_size(2 * n * sizeof(T))), n);
  }

  void *data() const { return (void *)storage.data(); }
  size_t size() const { return storage.size(); }

private:
  alignas(64) std::vector<std::byte> storage;
};

template <typename T>
struct SRef<T, location::device> {
  std::span<T> x, y, z;

  struct Ref {
    T &x, &y, &z;
  };

  struct RefConst {
    const T &x, &y, &z;
  };

  constexpr static size_t alignment = 128;

  __HOST_DEVICE__ Ref operator[](size_t i) { return {x[i], y[i], z[i]}; }

  __HOST_DEVICE__ const RefConst operator[](size_t i) const { return {x[i], y[i], z[i]}; }

  // Helper function to compute aligned size
  __HOST_DEVICE__ constexpr inline size_t align_size(size_t size) const {
    return ((size + alignment - 1) / alignment) * alignment;
  }

  SRef(size_t n) {
    CHECK_CUDA_ERROR(hipMalloc(&storage, align_size(3 * n * sizeof(T))));
    x = std::span<T>(reinterpret_cast<T *>(storage), n);
    y = std::span<T>(reinterpret_cast<T *>(storage) + align_size(n * sizeof(T)), n);
    z = std::span<T>(reinterpret_cast<T *>(storage) + align_size(2 * n * sizeof(T)), n);
  }

  void *data() const { return storage; }

private:
  alignas(64) void *storage;
};

template <typename T>
__global__ void MulTwoAoS(SRef<T, location::device> s) {
  auto tid = threadIdx.x + blockIdx.x * blockDim.x;
  if (tid < container_size) { s[tid].x = tid * 2; }
}

template <typename T>
__global__ void MulTwoSoA(SRef<T, location::device> s) {
  auto tid = threadIdx.x + blockIdx.x * blockDim.x;
  if (tid < container_size) { s.x[tid] = tid * 2; }
}

////////////////////////////////////////////////////////////////////

template <typename T>
static void BM_ReferencesAoSGPU(benchmark::State &state) {
  SRef<T, location::host> h_out(container_size);
  SRef<T, location::device> d_in(container_size);

  for (const auto &&_ : state) {
#ifdef USE_CUDA_EVENTS
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
#else
    hipDeviceSynchronize();
    auto start = Clock::now();
#endif

    MulTwoAoS<<<ceil(container_size / 256.), 256>>>(d_in);

#ifdef USE_CUDA_EVENTS
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    state.SetIterationTime(milliseconds / 1000.);
#else
    hipDeviceSynchronize();
    auto end = Clock::now();
    auto elapsed_seconds = std::chrono::duration_cast<fsecs>(end - start);
    state.SetIterationTime(elapsed_seconds.count());
#endif

    CHECK_CUDA_ERROR(hipGetLastError());
    state.counters["size"] = container_size;
  }

  CHECK_CUDA_ERROR(hipMemcpy(h_out.data(), d_in.data(), h_out.size(), hipMemcpyDeviceToHost));
  int i = 0;
  if (!std::all_of(h_out.x.begin(), h_out.x.end(), [&](int val) { return 2 * i++ == val; })) {
    state.SkipWithError("Incorrect result!");
    for (size_t j = 0; j < 10 && j < h_out.x.size(); ++j) {
      std::cout << h_out.x[j] << " ";
    }
    std::cout << std::endl;
  }

  hipFree(d_in.data());
}

template <typename T>
static void BM_ReferencesSoAGPU(benchmark::State &state) {
  SRef<T, location::host> h_out(container_size);
  SRef<T, location::device> d_in(container_size);

  for (const auto &&_ : state) {
#ifdef USE_CUDA_EVENTS
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
#else
    hipDeviceSynchronize();
    auto start = Clock::now();
#endif

    MulTwoSoA<<<ceil(container_size / 256.), 256>>>(d_in);

#ifdef USE_CUDA_EVENTS
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);
    state.SetIterationTime(milliseconds / 1000.);
#else
    hipDeviceSynchronize();
    auto end = Clock::now();
    auto elapsed_seconds = std::chrono::duration_cast<fsecs>(end - start);
    state.SetIterationTime(elapsed_seconds.count());
#endif
    CHECK_CUDA_ERROR(hipGetLastError());

    state.counters["size"] = container_size;
  }

  CHECK_CUDA_ERROR(hipMemcpy(h_out.data(), d_in.data(), h_out.size(), hipMemcpyDeviceToHost));
  int i = 0;
  if (!std::all_of(h_out.x.begin(), h_out.x.end(), [&](int val) { return 2 * i++ == val; })) {
    state.SkipWithError("Incorrect result!");
    for (size_t j = 0; j < 10 && j < h_out.x.size(); ++j) {
      std::cout << h_out.x[j] << " ";
    }
    std::cout << std::endl;
  }

  hipFree(d_in.data());
}

#define BENCHMARK_ARGS Unit(benchmark::kMillisecond)->UseManualTime();

BENCHMARK_TEMPLATE(BM_ReferencesAoSGPU, double)->BENCHMARK_ARGS;
BENCHMARK_TEMPLATE(BM_ReferencesSoAGPU, double)->BENCHMARK_ARGS;

BENCHMARK_MAIN();
